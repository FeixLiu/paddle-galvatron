#include "hip/hip_runtime.h"
// Copyright (c) 2025 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"
#include "all_reduce.cuh"

// Fake pointer type, must match fptr_t type in ops.h.
// We use this type alias to indicate when pointers are passed in as int64_t.
using fptr_t = int64_t;
static_assert(sizeof(void*) == sizeof(fptr_t));

fptr_t init_custom_all_reduce(const std::vector<fptr_t>& fake_ipc_ptrs,
                      paddle::Tensor& rank_data, int64_t rank,
                      bool full_nvlink) {
  int world_size = fake_ipc_ptrs.size();
  if (world_size > 8)
    throw std::invalid_argument("world size > 8 is not supported");
  if (world_size % 2 != 0)
    throw std::invalid_argument("Odd num gpus is not supported for now");
  if (rank < 0 || rank >= world_size)
    throw std::invalid_argument("invalid rank passed in");

  paddle::Signal* ipc_ptrs[8];
  for (int i = 0; i < world_size; i++) {
    ipc_ptrs[i] = reinterpret_cast<paddle::Signal*>(fake_ipc_ptrs[i]);
  }
  return (fptr_t) new paddle::CustomAllreduce(ipc_ptrs, rank_data.data(),
                                            rank_data.numel(), rank, world_size,
                                            full_nvlink);
}

/**
 * Performs an out-of-place allreduce and stores result in out.
 *
 * If _reg_buffer is null, assumes inp.data() is already IPC-registered.
 * Otherwise, _reg_buffer is assumed to be IPC-registered and inp is first
 * copied into _reg_buffer.
 */
void all_reduce(fptr_t _fa, paddle::Tensor& inp, paddle::Tensor& out,
                fptr_t _reg_buffer, int64_t reg_buffer_sz_bytes) {
  auto fa = reinterpret_cast<paddle::CustomAllreduce*>(_fa);
  auto stream = inp.stream();

  auto input_size = inp.numel() * 2;
  auto reg_buffer = reinterpret_cast<void*>(_reg_buffer);
  if (reg_buffer) {
    hipMemcpyAsync(reg_buffer, inp.data(), input_size,
                                  hipMemcpyDeviceToDevice, stream);
  } else {
    reg_buffer = inp.data();
  }
  switch (out.dtype()) {
    case phi::DataType::FLOAT32: {
      fa->allreduce<float>(stream, reinterpret_cast<float*>(reg_buffer),
                           reinterpret_cast<float*>(out.data()),
                           out.numel());
      break;
    }
    case phi::DataType::FLOAT16: {
      fa->allreduce<half>(stream, reinterpret_cast<half*>(reg_buffer),
                          reinterpret_cast<half*>(out.data()), out.numel());
      break;
    }
    case phi::DataType::BFLOAT16: {
      fa->allreduce<hip_bfloat16>(
          stream, reinterpret_cast<hip_bfloat16*>(reg_buffer),
          reinterpret_cast<hip_bfloat16*>(out.data()), out.numel());
      break;
    }
    default:
      throw std::runtime_error(
          "custom allreduce only supports float32, float16 and bfloat16");
  }
}

void dispose(fptr_t _fa) {
  delete reinterpret_cast<paddle::CustomAllreduce*>(_fa);
}

int64_t meta_size() { return sizeof(paddle::Signal); }

void register_buffer(fptr_t _fa, const std::vector<fptr_t>& fake_ipc_ptrs) {
  auto fa = reinterpret_cast<paddle::CustomAllreduce*>(_fa);
  void* ipc_ptrs[8];
  for (int i = 0; i < fake_ipc_ptrs.size(); i++) {
    ipc_ptrs[i] = reinterpret_cast<void*>(fake_ipc_ptrs[i]);
  }
  fa->register_buffer(ipc_ptrs);
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
std::tuple<std::vector<int64_t>, std::vector<int64_t>>
get_graph_buffer_ipc_meta(fptr_t _fa) {
  auto fa = reinterpret_cast<paddle::CustomAllreduce*>(_fa);
  auto [handle, offsets] = fa->get_graph_buffer_ipc_meta();
  std::vector<int64_t> bytes(handle.begin(), handle.end());
  return std::make_tuple(bytes, offsets);
}

// Use vector<int64_t> to represent byte data for python binding compatibility.
void register_graph_buffers(fptr_t _fa,
                            const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets) {
  auto fa = reinterpret_cast<paddle::CustomAllreduce*>(_fa);
  std::vector<std::string> bytes;
  bytes.reserve(handles.size());
  for (int i = 0; i < handles.size(); i++) {
    bytes.emplace_back(handles[i].begin(), handles[i].end());
  }
  bytes.reserve(handles.size());
  fa->register_graph_buffers(bytes, offsets);
}
